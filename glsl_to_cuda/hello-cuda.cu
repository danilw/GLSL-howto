#include "hip/hip_runtime.h"
#include <stdio.h>

//
// very simple GLSL to CUDA importing
// look for EXAMPLE 1 and EXAMPLE 2 in code

// build
// nvcc hello-cuda.cu -o hello-cuda
//

const unsigned xSize = 1024, ySize = 768; //out image size
//https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications__technical-specifications-per-compute-capability
//to support "big image resolution" need use loop for cuda_kernel calls, base on hipOccupancyMaxPotentialBlockSize
//if you have black image output, set block size to 16,16 and 256x256 image resolution

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)

__host__ __device__ inline int index(const int x, const int y) {
    return y * xSize + x;
}

//bmp image

void write_bmp(float *R, float *G, float *B) {
    FILE *f;
    unsigned char *img = NULL;
    unsigned int w = xSize, h = ySize, x, y;
    unsigned char r, g, b;
    int filesize = 54 + 3 * w*h; //w is your image width, h is image height, both int

    img = (unsigned char *) malloc(3 * w * h);
    memset(img, 0, 3 * w * h);

    for (int i = 0; i < w; i++) {
        for (int j = 0; j < h; j++) {
            x = i;
            y = (h - 1) - j;
            r = (unsigned char) (R[index(i, j)]*255);
            g = (unsigned char) (G[index(i, j)]*255);
            b = (unsigned char) (B[index(i, j)]*255);
            img[(x + y * w)*3 + 2] = r;
            img[(x + y * w)*3 + 1] = g;
            img[(x + y * w)*3 + 0] = b;
        }
    }

    unsigned char bmpfileheader[14] = {'B', 'M', 0, 0, 0, 0, 0, 0, 0, 0, 54, 0, 0, 0};
    unsigned char bmpinfoheader[40] = {40, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 24, 0};
    unsigned char bmppad[3] = {0, 0, 0};

    bmpfileheader[ 2] = (unsigned char) (filesize);
    bmpfileheader[ 3] = (unsigned char) (filesize >> 8);
    bmpfileheader[ 4] = (unsigned char) (filesize >> 16);
    bmpfileheader[ 5] = (unsigned char) (filesize >> 24);

    bmpinfoheader[ 4] = (unsigned char) (w);
    bmpinfoheader[ 5] = (unsigned char) (w >> 8);
    bmpinfoheader[ 6] = (unsigned char) (w >> 16);
    bmpinfoheader[ 7] = (unsigned char) (w >> 24);
    bmpinfoheader[ 8] = (unsigned char) (h);
    bmpinfoheader[ 9] = (unsigned char) (h >> 8);
    bmpinfoheader[10] = (unsigned char) (h >> 16);
    bmpinfoheader[11] = (unsigned char) (h >> 24);

    f = fopen("img.bmp", "wb");
    fwrite(bmpfileheader, 1, 14, f);
    fwrite(bmpinfoheader, 1, 40, f);
    for (int i = 0; i < h; i++) {
        fwrite(img + (w * (h - i - 1)*3), 3, w, f);
        fwrite(bmppad, 1, (4 - (w * 3) % 4) % 4, f);
    }

    free(img);
    fclose(f);
}

//--------------------------------
//EXAMPLE 1

//base on https://www.shadertoy.com/view/llXSD8
//shader source
/*
void mainImage(out vec4 f, vec2 u) {
    f -= f;
    u *= 3. / iResolution.y;
    for (float i = -2.; i <= 1.; i += .1)
        f += (i * i + i + 1.) / 3e2 / abs(i * (u.y - u.x - i) - u.x + 2.);
    f *= abs(sin(u.y * 1e2));
    f.rb *= 0.;
}
 */

//uncomment this to use
/*
__global__ void mainImage(float* R, float* G, float* B) {

    //i,j its fragCoord.xy
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    float u[2];
    float iResolution[2];
    u[0] = i;
    u[1] = j;
    iResolution[0] = xSize;
    iResolution[1] = ySize;

    u[0] *= 3. / iResolution[1];
    u[1] *= 3. / iResolution[1];
    float f; //single channel color
    for (float i_ = -2.; i_ <= 1.; i_ += .1)
        f += (i_ * i_ + i_ + 1.) / 3e2 / abs(i_ * (u[1] - u[0] - i_) - u[0] + 2.);
    f=min(max(f,0.),1.); //for image, max color 1
    f *= abs(sin(u[1] * 1e2));

    R[index(i, j)] = 0.;
    G[index(i, j)] = f;
    B[index(i, j)] = 0.;

}
 */
//--------------------------------


//--------------------------------
//EXAMPLE 2

//base on https://www.shadertoy.com/view/lsd3zr
//shader source
/*
#define L    +u ; v = 1.- v*v; f += .02/min(v.x,v.y);
#define S(c) v = c.5 L    v = vec2(u.x c-2.,c 1.-u.x) L

void mainImage( out vec4 f, vec2 u ) {
    f-=f;
    vec2 v = iResolution.xy;
    u = (u+u-v)/v.y/.5;
    S() S(-)
}
 */
/**/
#define device_vec(val) for (int j_ = 0; j_ < val; j_++)
#define L    +u[j_] ; device_vec(2)v[j_] = 1.- v[j_]*v[j_]; f += .02/min(v[0],v[1]);
#define S(c) device_vec(2)v[j_] = c.5 L v[0] = u[0] c-2.;v[1]=c 1.-u[0] ; device_vec(2)v[j_] +=L

__global__ void mainImage(float* R, float* G, float* B) {

    //i,j its fragCoord.xy
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    float u[4];
    float v[2];
    u[0] = i;
    u[1] = j;
    v[0] = xSize;
    v[1] = ySize;
    float f = 0.; //single channel color
    device_vec(2) u[j_] = (u[j_] + u[j_] - v[j_]) / v[1] / .5;
    S() S(-)

    f = min(max(f, 0.), 1.); //for image, max color 1

    R[index(i, j)] = f;
    G[index(i, j)] = f;
    B[index(i, j)] = f;

}

//--------------------------------

int main() {
    const dim3 blockSize(16, 16); //16,16 for 256x256 image, can be 32,32 for 1024x768
    const dim3 gridSize(xSize / 16, ySize / 16);

    float *R_host, *G_host, *B_host;
    float *R_dev, *G_dev, *B_dev;
    hipHostMalloc((void**) &R_host, xSize * ySize * sizeof (float));
    cudaCheckErrors("Failed to allocate host buffer.");
    hipHostMalloc((void**) &G_host, xSize * ySize * sizeof (float));
    cudaCheckErrors("Failed to allocate host buffer.");
    hipHostMalloc((void**) &B_host, xSize * ySize * sizeof (float));
    cudaCheckErrors("Failed to allocate host buffer.");


    hipMalloc((void**) &R_dev, xSize * ySize * sizeof (float));
    cudaCheckErrors("Failed to allocate device buffer.");
    hipMalloc((void**) &G_dev, xSize * ySize * sizeof (float));
    cudaCheckErrors("Failed to allocate device buffer.");
    hipMalloc((void**) &B_dev, xSize * ySize * sizeof (float));
    cudaCheckErrors("Failed to allocate device buffer.");



    hipMemcpy(R_dev, R_host, xSize * ySize * sizeof (float), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");
    hipMemcpy(G_dev, G_host, xSize * ySize * sizeof (float), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    mainImage << <gridSize, blockSize>>>(R_dev, G_dev, B_dev);
    cudaCheckErrors("Kernel launch failure");


    hipMemcpy(R_host, R_dev, xSize * ySize * sizeof (float), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");
    hipMemcpy(G_host, G_dev, xSize * ySize * sizeof (float), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");
    hipMemcpy(B_host, B_dev, xSize * ySize * sizeof (float), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    write_bmp(R_host, G_host, B_host);

    hipFree(R_dev);
    cudaCheckErrors("hipFree fail");
    hipFree(G_dev);
    cudaCheckErrors("hipFree fail");
    hipFree(B_dev);
    cudaCheckErrors("hipFree fail");
}
